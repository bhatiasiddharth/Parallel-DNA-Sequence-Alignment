#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#define threads 8


/* 
~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~README~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
1. To run the code : 
    
   nvcc -deviceemu cuda1.cu  (in emulation mode)
   a.exe 8                   (exefile sizeofstring)


2. Optimizations not performed 
3. Configuration for each kernel function will change according to size
4. Code not tested for various other sizes
5. Parallel max algo may be slower than expected

*/



/* 
~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~CODE~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
kernel functions
1 : creation of substitution matrix (H)    ( Time complexity : O( Nlog(N) )  )
2 : Finding maximum in H matrix            ( Time complexity : O( Nlog(N) )  )
3 : creation of bactrace matrix (pr_dest)  (stores destination location from each cell for performing bactracking )   ( Time complexity : O( N )  )

sequential code : Finds aligned sequences (backtracking)    ( Time complexity : O( N )  )
*/



//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~  1 : creation of H matrix  ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~// 

__global__ void matrix_num(char *X,char *Y,int* FF, int* HH1, int* HH ,int* EE,int N , int i, int Ge, int Gs, int S, int D)
{    
    //find Row and Column corresponding to a data element for each thread
    int loc = blockIdx.y * blockDim.y + threadIdx.y;

	int inloc = i * (N+1) + loc + 1;		//value to be updated
	

	///// done with FF ////
	loc = (i-1)*(N+1)+loc+1;	
	
	FF[inloc] = FF[loc] - Ge ;
	if(HH[loc] - Gs - Ge  > FF[inloc])
		FF[inloc] = HH[loc] - Gs - Ge ;
		
	///// done with HH1 ///
	if( X[i-1]==Y[blockIdx.y * blockDim.y + threadIdx.y ] )
		HH1[ inloc ] = HH[ loc-1 ] + S ; 	// loc-1 is diagonal above
	else 
		HH1[ inloc ] = HH[ loc-1 ] + D ;  
	
	if( HH1[ inloc ] < 0 ) HH1[ inloc ] = 0 ;
	if( HH1[ inloc ] < FF[inloc] )  HH1[ inloc ] = FF[inloc] ;	
	EE[inloc]  = HH1[inloc] ;
}

__global__ void matrix_num3(int* HH1, int* EE, int* HH, int N , int i, int Gs)
{
	int inloc = i*(N+1) + blockIdx.y * blockDim.y + threadIdx.y + 1;
	HH[inloc] = HH1[inloc]  ;
	if( HH[inloc] < EE[inloc] - Gs )
		HH[inloc] = EE[inloc] - Gs ; 	
}

__global__ void prefixsum(int* HH1, int* EE , int index ,int N, int Ge){
	
	/// loc+1 is the memory

	int k,loc = index*(N+1)+blockIdx.y * blockDim.y + threadIdx.y, j ;
	int temp ;

	j=N;

	/// upsweep
	for(j = 2; j < N; j*=2){
		if( (loc % j) == 0 && (loc+1+j-1)/(N+1)==(loc)/(N+1) )
			EE[loc+1+j-1] = (EE[loc+1+j/2-1] > EE[loc+1+j-1] + Ge * j/2) ? EE[loc+1+j/2-1] : EE[loc+1+j-1] + Ge * j/2 ;
	}

	if( (loc+1)%(N+1) == N ){
    	EE[loc+1] = 0 ;	
    }

}

__global__ void prefixsum2(int* HH1, int* EE , int index ,int N, int Ge){

	int k,loc = index*(N+1)+blockIdx.y * blockDim.y + threadIdx.y, j = N;
	int temp ;    
	

    /// downsweep
	for(j = j ; j > 1 ; j/=2 ) {
		if((loc % j) == 0)  {   //// intially the first thread works 

			temp = EE[loc + 1 + j/2 -1];
			EE[loc + 1 + j/2 -1] = EE[loc + 1 + j - 1];
			EE[loc + 1 + j - 1] = ((temp > EE[loc + 1 + j - 1]) ? temp  : EE[loc + 1 + j - 1]) -	  Ge * j/2;
		} 
	}

}



//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ 2 : parallel max ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//

__global__ void findStore(int* HH , int* store , int N , int i)
{
	int locx = blockIdx.x * blockDim.x + threadIdx.x+1 ;
	int locy = blockIdx.y * blockDim.y + threadIdx.y+1 ;

	store[ locy*(N+1) + locx ] = 0 ;

	if(  HH[ i*(N+1)+locy ] >=  HH[ i*(N+1)+locx ]  ) store[locy*(N+1)+locx] = 1 ;
		
}

__global__ void initAnd(int* array , int N)
{
	array[ blockIdx.y * blockDim.y + threadIdx.y+1] = 1 ;
}

__global__ void parAnding(int* store , int* array , int N)
{
	int locx = blockIdx.x * blockDim.x + threadIdx.x+1 ;
        int locy = blockIdx.y * blockDim.y + threadIdx.y+1 ;

	array[locy] = array[locy] & store[ locy*(N+1)+locx ] ; 
}

__global__ void giveMax(int* HH,int* array,int N, int i,int *maxX, int *maxY)
{
	int locy = blockIdx.y * blockDim.y + threadIdx.y+1 ;
	if( array[locy]==1 && HH[i*(N+1)+locy]  > HH[*maxY * (N+1) + *maxX] ) { *maxX = locy ; *maxY = i ; }  
}



//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ 3 : back trace ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//


__global__ void traceback(char*XX , char* YY , int* HH , int* dest , int N , int S, int D, int Gs, int Ge)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	int k ;
	
	int loc=i*(N+1)+j;
	dest[loc]=0 ;

	if(i>0 && j>0){

		if( XX[i-1]==YY[j-1] && HH[(i-1)*(N+1)+j-1]+S == HH[loc] ) dest[loc] = (i-1)*(N+1)+j-1 ;
	
		else if( XX[i-1]!=YY[j-1] && HH[(i-1)*(N+1)+j-1]+D == HH[loc] ) dest[loc] = (i-1)*(N+1)+j-1 ;

		else {

			for( k=i-1; k>=0 ; k-- ) {
				if ( HH[ k*(N+1)+j ] - Gs - (i-k)*Ge   == HH[loc] ) { dest[loc] = k*(N+1)+j ; break ; }
			}

			for( k=j-1; k>=0 ; k--) {
				if ( HH[ i*(N+1)+k ] - Gs - (j-k)*Ge == HH[loc] ) { dest[loc] = i*(N+1)+k ; break ; }
			} 
		}

 	}

}





//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~  MAIN FUNCTION ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//



int main(int argc, char *argv[])
{
	int N, i, j;				//N == size of square matrix
	int Ge=1, Gs=8, S=5, D=-3;
	N = atoi(argv[1]);	
	
	int *F,*FF,*H1,*HH1,*E,*EE,*H,*HH,*dest,*store,*array;
	char *X,*Y,*a,*b,*seq1,*seq2;

	int *pr_dest ;

	size_t size=sizeof(int)* (N+1) * (N+1);
	size_t size_str= sizeof(char)* N ;
	size_t sizeN=sizeof(int)* (N+1) ;

	freopen("oupt.txt","w",stdout);
	
    //allocate host side memory
	a=(char*)malloc(size_str);
	b=(char*)malloc(size_str);
	seq1=(char*)malloc(size_str);
	seq2=(char*)malloc(size_str);
	F=(int*)malloc(size);
	H1=(int*)malloc(size);
	E=(int*)malloc(size);
	H=(int*)malloc(size);

	//dest=(int*)malloc(size);
	pr_dest=(int*)malloc(size);
	//store=(int*)malloc(size);
	//array=(int*)malloc(size);


	int *maxXX = 0, *maxYY=0, *maxX = 0, *maxY = 0 ;
	maxX = (int*) calloc(1, sizeof(int));
	maxY = (int*) calloc(1, sizeof(int));
	hipMalloc(&maxXX, sizeof(int));
	hipMalloc(&maxYY, sizeof(int));
	hipMemcpy(maxXX, maxX, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(maxYY, maxY, sizeof(int), hipMemcpyHostToDevice);

	FILE *fp = fopen("seq.txt","r") ;
	
	fgets(a,20000,fp) ;
	fgets(b,20000,fp) ;


    //allocate device memory
	hipMalloc(&X,size_str);  //	printf("\nAfter cudaMalloc for X\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&Y,size_str);  //	printf("\nAfter cudaMalloc for Y\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&FF,size);     //	printf("\nAfter cudaMalloc for FF\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&HH1,size);    //	printf("\nAfter cudaMalloc for HH1\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&EE,size);     //	printf("\nAfter cudaMalloc for EE\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&HH,size);     //	printf("\nAfter cudaMalloc for HH\t%s\n",cudaGetErrorString(cudaGetLastError()));
    hipMalloc(&dest,size);   //    printf("\nAfter cudaMalloc for HH\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&store,size);  //    printf("\nAfter cudaMalloc for HH\t%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&array,sizeN); //    printf("\nAfter cudaMalloc for HH\t%s\n",cudaGetErrorString(cudaGetLastError()));


	
	for(i=0;i<=N;i++){   for(j=0;j<=N;j++){  F[i*(N+1)+j]=0;  H1[i*(N+1)+j]=0;  E[i*(N+1)+j]=0;  H[i*(N+1)+j]=0;  }  }
		
    hipMemcpy(X,a,size_str,hipMemcpyHostToDevice);
	hipMemcpy(Y,b,size_str,hipMemcpyHostToDevice);
 	hipMemcpy(FF,F,size,hipMemcpyHostToDevice);
 	hipMemcpy(HH1,H1,size,hipMemcpyHostToDevice);
 	hipMemcpy(EE,E,size,hipMemcpyHostToDevice);
 	hipMemcpy(HH,H,size,hipMemcpyHostToDevice);

//	printf("\nAfter HostToDevice Memcpy\n%s\n",cudaGetErrorString(cudaGetLastError()));








    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ calculate execution configuration ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//
	    	
	dim3 blocksize(1,threads);		        //each block contains 16 * 16 (=256) threads 
	int k = (N/threads) + (N%threads != 0);
	dim3 gridsize(1,k);			//creating just sufficient no of blocks
    

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ kernel call 1 : H-matrix calculation ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~// 
    //GPU timer code
   	
   	float time1;
   	hipEvent_t start1,stop1;			
	hipEventCreate(&start1);		
	hipEventCreate(&stop1);
	hipEventRecord(start1,0);

	/////for(i=0;i<=N;i++){for(j=0;j<=N;j++){ H1[i*(N+1)+j]=5; }} cudaMemcpy(HH1,H1,size,cudaMemcpyHostToDevice);
	
	for(i=1;i<=N;i++)
	{
		matrix_num <<< gridsize, blocksize >>> (X, Y, FF, HH1,HH,EE, N, i, Ge, Gs, S, D);
		prefixsum <<< gridsize, blocksize >>> (HH1, EE ,i, N, Ge); 
		prefixsum2 <<< gridsize, blocksize >>> (HH1, EE ,i, N, Ge); 	
		matrix_num3 <<< gridsize, blocksize >>>  (HH1 , EE , HH ,N ,i ,Gs );	
	}

	hipEventRecord(stop1,0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&time1,start1,stop1);			//time taken in kernel call calculated
	hipEventDestroy(start1);
	hipEventDestroy(stop1);




    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ calculate execution configuration ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//
   
    dim3 block(threads,threads);                      //each block contains 16 * 16 (=256) threads
    k = (N/threads) + (N%threads != 0);
    dim3 grid(k,k);        


    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ kernel call 2 : Parallel Max calculation ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~// 
    //GPU timer code
   	
   	float time2;
   	hipEvent_t start2,stop2;			
	hipEventCreate(&start2);		
	hipEventCreate(&stop2);
	hipEventRecord(start2,0);

	for(i=1;i<=N;i++)
	{
		findStore  <<< grid, block >>> (HH, store, N, i);
		initAnd <<< gridsize, blocksize >>> (array , N );
		parAnding <<< grid, block >>> (store , array , N) ;
		giveMax   <<< gridsize, blocksize >>> (HH , array , N ,i , maxXX , maxYY) ; 	

	}

	hipEventRecord(stop2,0);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&time2,start2,stop2);			//time taken in kernel call calculated
	hipEventDestroy(start2);
	hipEventDestroy(stop2);

	//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ kernel call 3 : Traceback Matrix calculation ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~// 
    //GPU timer code

    float time3;
   	hipEvent_t start3,stop3;			
	hipEventCreate(&start3);		
	hipEventCreate(&stop3);
	hipEventRecord(start3,0);

    traceback <<< grid , block >>> (X,Y,HH,dest,N,S,D,Gs,Ge);

    hipEventRecord(stop3,0);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&time3,start3,stop3);			//time taken in kernel call calculated
	hipEventDestroy(start3);
	hipEventDestroy(stop3);

	//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//
	//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//



	hipMemcpy(E,EE,size,hipMemcpyDeviceToHost);
	hipMemcpy(H,HH,size,hipMemcpyDeviceToHost);		
	hipMemcpy(pr_dest,dest,size,hipMemcpyDeviceToHost);	
	hipMemcpy(maxX, maxXX, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(maxY, maxYY, sizeof(int), hipMemcpyDeviceToHost);






    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Backtrace ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//

	int i1,j1,t,l,count;
    i=*maxY; j=*maxX; t=0; l=0;

    while(i>0 && j>0) {
        i1=pr_dest[i*(N+1)+j]/(N+1);
        j1=pr_dest[i*(N+1)+j]%(N+1);
 		
        if((i-i1) > (j-j1) ) {
            count = j1-i1;
            while(count-- > 0){
                seq1[l++] = a[i1];
                seq2[t++] = '-';
            }
        }
	
		else if(i-i1 < j-j1){
            count = i1-j1;
            while(count-- > 0){
                seq2[t++] = b[j1];
                seq1[l++] = '-';
            }
        }
	
	    else{			
            seq1[l++]=a[i1];
            seq2[t++]=b[j1];
        }

        i= i1;
        j= j1;
    }
	

//	while(j>=0){   seq2[t++]=b[j--];   seq1[l++]='-';   }
//  while(i>=0){   seq1[l++]=a[i--];   seq2[t++]='-';   }





//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Output ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//


	printf("Substitution Matrix \n\n");

	for(i=0;i<=N;i++)
	{
		for(j=0;j<=N;j++){
			printf("%d\t",H[i*(N+1)+j]);
		}
		printf("\n");
	}


    printf("\n\nBacktrack Path Matrix \n\n");
	for(i=0;i<=N;i++){
        for(j=0;j<=N;j++){
            printf("%d\t",H[i*(N+1)+j]);
            printf("(%d,%d)\t",     pr_dest[i*(N+1)+j]/(N+1) , pr_dest[i*(N+1)+j]%(N+1) );
        }
        printf("\n");
    }



    printf("\n\nOriginal Sequences\n\n");
	for(j=0;j<N;j++){
        printf("%c",a[j]);
    }
    printf("\n");

    for(j=0;j<N;j++){
        printf("%c",b[j]);
    }
    printf("\n");
  
    printf("\n\nMaximally locally Alligned Sequence\n\n");
	for(j=l-1;j>=0;j--){
        printf("%c",seq1[j]);
    }
    printf("\n");

    for(j=t-1;j>=0;j--){
        printf("%c",seq2[j]);
    }
    printf("\n");

    printf("\n\nMaximum\n");
    printf("\n%d %d H[x][y]=%d\n",*maxY,*maxX,H[*maxY*(N+1)+*maxX]);


    printf("\nTime taken :\n");
    printf("Time to find substitution matrix = %f (ms)\n",time1);
    printf("Time to find maximum = %f (ms)\n",time2);
    printf("Time to find traceback = %f (ms)\n",time3);


    
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ DEBUGGING ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ PREFIX SUM DEBUGGING ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//


	
	//printf("\n");
	//H1[0]=0 ;  H1[1]=2 ; H1[2]=3 ; H1[3]=4 ; H1[4]=5 ; H1[5]=6 ; H1[6]=7 ; H1[7]=8 ; H1[8]=9 ; cudaMemcpy(HH1,H1,size,cudaMemcpyHostToDevice);
	//E[0]=0 ;  E[1]=2 ; E[2]=3 ; E[3]=4 ; E[4]=5 ; E[5]=6 ; E[6]=7 ; E[7]=8 ; E[8]=9 ; cudaMemcpy(EE,E,size,cudaMemcpyHostToDevice);
	//E[0]=0 ;  E[1]=2 ; E[2]=4 ; E[3]=4 ; E[4]=8 ; E[5]=6 ; E[6]=8 ; E[7]=8 ; E[8]=0 ; cudaMemcpy(EE,E,size,cudaMemcpyHostToDevice);
	//for(i=0;i<=N;i++) { printf( "%d\t",E[i] ) ; } printf("\n");

	//prefixsum <<< gridsize, blocksize >>> (HH1, EE , 0 , N, Ge); 
	//prefixsum2 <<< gridsize, blocksize >>> (HH1, EE , 0, N, Ge); 	
   	//cudaMemcpy(E,EE,size,cudaMemcpyDeviceToHost);
   	//for(i=0;i<=N;i++) { printf( "%d\t",E[i] ) ; }

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~//	

    


    hipFree(FF); hipFree(HH1); hipFree(EE) ; hipFree(HH); hipFree(X); hipFree(Y); hipFree(array); hipFree(store); hipFree(dest);	



    return 0;
}


